
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define error 1e-6
#define BLOCK_SIZE 32

void init_Array(float *x, int m, int n){
    for(int i = 0; i < m; i++){
        for(int j = 0; j < n; j++){
            x[i * n + j] = rand() % 1000;
        }
    } 
}

void init_0_Array(float *x, int row, int col){
    bool ultimo =  false;

    for(int i = 0; i < row; i++){
        for(int j = 0; j < col; j++){

            if(j ==  col - 1 && !ultimo){
                ultimo = true;
            }
            float r = (float) rand() / RAND_MAX;
            if(r <= 0.25){

                x[i*col + j] = rand() % 1000;
                ultimo = false;

            } else{

                if (!ultimo){

                    x[i*col + j] = 0;

                } else {

                    x[i*col + j] = rand() % 1000;
                    ultimo = false;

                }
            }
        }
    }

}

int count_Num(float *x, int row, int col){

    int count = 0;

    for(int i = 0; i < row; i++){
        for(int j = 0; j < col; j++){
            if(x[i*col + j] != 0){
                count++;
            }
        }
    }

    return count;
}


void sparse_matrix(float *x, int *rows, int *cols, float *val, int row, int col){

    int antes = -1;
    int count = 0;
    int count_row = 0;
    for(int i = 0; i < row; i++){
        for(int j = 0; j < col; j++){
            if(x[i*col + j] != 0){

                if(i != antes){
                    rows[count_row] = count;
                    antes = i;
                    count_row++;
                }
                cols[count] = j;
                val[count] = x[i*col + j];
                count++;
            }
        }
    }
}

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

void print_matrix(float *a, int row, int col){

    for(int i = 0; i < row; i++){
        for(int j = 0; j < col; j++){
            printf("%f ",a[i * col + j]);
        }
        printf("\n");
    }
    printf("\n");
}

void print_imatrix(int *a, int row, int col){

    for(int i = 0; i < row; i++){
        for(int j = 0; j < col; j++){
            printf("%d ",a[i * col + j]);
        }
        printf("\n");
    }
    printf("\n");
}
__global__ void mmatrix(float *a, float *b, float *c, int m, int n, int k){

    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0;


    if(col < k && row < m){

        for(int i = 0; i < n; i++){
            sum += a[row * n + i] * b[i * k + col];
        }

        c[row * k + col] = sum;
    }
}

__global__ void gpuMatrixConv(float *a, float *b, float *c, int row1, int col1, int row2, int col2, int row3, int col3)
{
	unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0;

	if (row < row3 && col < col3) {
		for (int i = 0; i < row2; i++) {
			for (int j = 0; j < col2; j++) {
                sum += a[(row + i) * col1 + col + j] * b[i * row2 + j];
                
            }
        }
		c[row * col3 + col] = sum;
	}
}

__global__ void gpuMatrixTranpose(float *a, float *b, int rows, int cols){

    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < rows && col < cols){

        int pos_a = row * cols + col;
        int pos_b = col * rows + row;
        b[pos_b] = a[pos_a];
        
    }
}

__global__ void gpuMVSparse(float *values, float *vector, int *rows, int *cols,float *res, int row){

    unsigned int Id = threadIdx.x + blockDim.x * blockIdx.x;

    if(Id < row){
        for(int k = rows[Id]; k < rows[Id+1]; k++){
            res[Id] += values[k]*vector[cols[k]];
        }
    }
}

void sparse_preparation(){

    int col, row;
    bool bien = true;

    printf("\n");
    printf("\n");
    printf("Introduce las filas de A:\n");
    fflush(stdout);
    scanf("%d", &row);
    printf("Introduce las columnas de A:\n");
    fflush(stdout);
    scanf("%d", &col);


    double start_GPU, stop_GPU;
    double start_CPU, stop_CPU;
    double diferencia_CPU, diferencia_GPU;

    float *A = (float *)malloc(row * col * sizeof(float));
    float *vector = (float *)malloc(col * sizeof(float));

    init_Array(vector, 1, col);
    init_0_Array(A, row, col);
    int size = count_Num(A, row, col);

    float *values = (float *)malloc(size * sizeof(float));
    int *rows = (int *)malloc((row + 1) * sizeof(int));
    int *cols = (int *)malloc(size * sizeof(int));

    float *res = (float *)malloc(row * sizeof(float));
    float *res_F = (float *)malloc(row * sizeof(float));

    sparse_matrix(A, rows, cols, values, row, col);
    rows[row] = size;

    int *rows_GPU;
    int *cols_GPU;
    float *values_GPU;
    float *res_GPU;
    float *vector_GPU;

    hipMalloc(&rows_GPU, (row + 1) * sizeof(int));
    hipMalloc(&cols_GPU, size * sizeof(int));
    hipMalloc(&values_GPU, size * sizeof(float));
    hipMalloc(&res_GPU, row * sizeof(float));
    hipMalloc(&vector_GPU, col * sizeof(float));
    hipMemset(res_GPU, 0, row * sizeof(float));

    hipMemcpy(rows_GPU, rows, (row +1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cols_GPU, cols, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(values_GPU, values, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(vector_GPU, vector, col * sizeof(float), hipMemcpyHostToDevice);
    //Start CPU Part//

    start_CPU = cpuSecond();


    for(int k = 0; k < row; k++){
        res[k] = 0;
    }
    for(int i = 0; i < row; i++){
        for(int k = rows[i]; k < rows[i + 1]; k++){

            res[i] += values[k]*vector[cols[k]];
        }
    }

    stop_CPU = cpuSecond();
    diferencia_CPU = stop_CPU - start_CPU;
    

    //Stop Cpu Part // 

    unsigned int GRID = col + BLOCK_SIZE - 1 / BLOCK_SIZE;

    
    //Init GPU part//

    start_GPU = cpuSecond();

    gpuMVSparse<<<GRID, BLOCK_SIZE>>>(values_GPU, vector_GPU, rows_GPU, cols_GPU, res_GPU, row);

    hipDeviceSynchronize();
    hipMemcpy(res_F, res_GPU, row * sizeof(float), hipMemcpyDeviceToHost);

    stop_GPU = cpuSecond();

    diferencia_GPU = stop_GPU - start_GPU;

    //Stop GPU part//

    //Start Checking //
    
    for(int j = 0; j < row; j++){
        if(fabs(res_F[j] - res[j]) >= error ){
            bien = false;
            printf("Error en: %f %f\n", res_F[j], res[j]);
        }
    }


    if(bien){
        printf("Comparing the output for each implementation.. Correct!\n");
    }else {
        printf("Comparing the output for each implementation.. Incorrect!\n");
    }

    char d;
    printf("Do you want to print the matrix:\n");
    printf("YES: y  or NO: n\n");
    fflush(stdout);
    scanf(" %c", &d);
    if(d == 'y'){
        print_matrix(A,row,col);
        print_matrix(values, 1, size);
        print_imatrix(rows, 1, (col + 1));
        print_imatrix(cols, 1 ,size);
        print_matrix(res,row,1);
        print_matrix(res_F,row,1);
        fflush(stdout);
    }

    printf("Duration of the CPU: %f\n", diferencia_CPU);
    printf("Duration of the GPU: %f\n", diferencia_GPU);

    delete[] A;
    delete[] vector;
    delete[] cols;
    delete[] rows;
    delete[] res;
    delete[] res_F;
    delete[] values;
    hipFree(values_GPU);
    hipFree(cols_GPU);
    hipFree(rows_GPU);
    hipFree(res_GPU);
    hipFree(vector_GPU);
    
}

void tranpose_preparation(){

    int col, row;
    bool bien = true;

    printf("\n");
    printf("\n");
    printf("Introduce las filas de A:\n");
    fflush(stdout);
    scanf("%d", &row);
    printf("Introduce las columnas de A:\n");
    fflush(stdout);
    scanf("%d", &col);


    double start_GPU, stop_GPU;
    double start_CPU, stop_CPU;
    double diferencia_CPU, diferencia_GPU;

    float *A = (float *)malloc(row * col * sizeof(float));
    float *res = (float *)malloc(row * col * sizeof(float));
    float *res_F = (float *)malloc(row * col * sizeof(float));



    float *A_GPU;
    float *res_GPU;

    hipMalloc(&A_GPU, row * col * sizeof(float));
    hipMalloc(&res_GPU, row  * col * sizeof(float));

    init_Array(A, row, col);

    hipMemcpy(A_GPU, A, row * col * sizeof(float), hipMemcpyHostToDevice);

    //Start CPU Part//

    start_CPU = cpuSecond();

    for(int i = 0; i < row; i++){
        for(int j = 0; j < col; j++){

            int pos_a = i * col + j;
            int pos_res = j * row + i;

            res[pos_res] = A[pos_a]; 
        }
    }


    stop_CPU = cpuSecond();
    diferencia_CPU = stop_CPU - start_CPU;
    //Stop Cpu Part // 

    unsigned int grid_rows = (row + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_colm = (col + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 dimGrid(grid_colm, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    
    //Init GPU part//

    start_GPU = cpuSecond();

    gpuMatrixTranpose<<<dimGrid, dimBlock>>>(A_GPU, res_GPU, row, col);

    hipDeviceSynchronize();
    hipMemcpy(res_F, res_GPU, row * col * sizeof(float), hipMemcpyDeviceToHost);

    stop_GPU = cpuSecond();

    diferencia_GPU = stop_GPU - start_GPU;

    //Stop GPU part//

    //Start Checking //

    for(int i = 0; i < row; i++){
        for(int j = 0; j < col; j++){
            if(fabs(res_F[i * col + j] - res[i*col + j]) >= error ){
                bien = false;
                printf("Error en: %f %f\n", res_F[i * col + j], res[i * col + j]);
            }
        }
    }

    if(bien){
        printf("Comparing the output for each implementation.. Correct!\n");
    }else {
        printf("Comparing the output for each implementation.. Incorrect!\n");
    }

    char d;
    printf("Do you want to print the matrix:\n");
    printf("YES: y  or NO: n\n");
    fflush(stdout);
    scanf(" %c", &d);
    if(d == 'y'){
        print_matrix(A,row,col);
        print_matrix(res,col,row);
        print_matrix(res_F,col,row);
        fflush(stdout);
    }

    printf("Duration of the CPU: %f\n", diferencia_CPU);
    printf("Duration of the GPU: %f\n", diferencia_GPU);

    hipHostFree(A);
    hipHostFree(res);
    hipHostFree(res_F);
    hipFree(A_GPU);
    hipFree(res_GPU);

}

void conv_preparation(){

    int col1, row1, col2, row2, col3, row3;
    bool bien = true;

    INTRO: printf("\n");
    printf("\n");
    printf("Introduce las filas de A:\n");
    fflush(stdout);
    scanf("%d", &row1);
    printf("Introduce las columnas de A:\n");
    fflush(stdout);
    scanf("%d", &col1);
    printf("Introduce las fila y columnas de B:\n");
    fflush(stdout);
    scanf("%d", &col2);
    row2 = col2;

    if(row2 >= row1 || col2 >= col1 ) {
        printf("Introduces mal los numeros, la matriz A debe ser mayor que B");
        goto INTRO;
    }

    col3 = col1 - col2 + 1;
    row3 = row1 - row2 + 1;

    double start_GPU, stop_GPU;
    double start_CPU, stop_CPU;
    double diferencia_CPU, diferencia_GPU;

    float *A = (float *)malloc(row1 * col1 * sizeof(float));
    float *B = (float *)malloc(row2 * col2 * sizeof(float));
    float *res = (float *)malloc(row3 * col3 * sizeof(float));
    float *res_F = (float *)malloc(row3 * col3 * sizeof(float));



    float *A_GPU;
    float *B_GPU;
    float *res_GPU;

    hipMalloc(&A_GPU, row1 * col1 * sizeof(float));
    hipMalloc(&B_GPU, row2 * col2 * sizeof(float));
    hipMalloc(&res_GPU, row3  * col3 * sizeof(float));

    init_Array(A, row1, col1);
    init_Array(B, row2, col2);

    hipMemcpy(A_GPU, A, row1 * col1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_GPU, B, row2 * col2 * sizeof(float), hipMemcpyHostToDevice);

    //Start CPU Part//

    start_CPU = cpuSecond();

    int i, j ,k, z;
    float sum = 0.0;

    for(i = 0; i < row3; i++){
        for(z = 0; z < col3; z++){

            sum = 0.0;

            for(j = 0; j < row2; j++){
                for(k = 0; k < col2; k++){

                    sum += A[(i + j) * col1 + z + k] * B[j * row2 + k];
                
                }
            }
            res[i * col3 + z] = sum;
        }
    }


    stop_CPU = cpuSecond();
    diferencia_CPU = stop_CPU - start_CPU;
    //Stop Cpu Part // 

    unsigned int grid_rows = (row3 + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_colm = (col3 + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 dimGrid(grid_colm, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    
    //Init GPU part//

    start_GPU = cpuSecond();

    gpuMatrixConv<<<dimGrid, dimBlock>>>(A_GPU, B_GPU, res_GPU, row1, col1, row2, col2, row3, col3);

    hipDeviceSynchronize();
    hipMemcpy(res_F, res_GPU, row3 * col3 * sizeof(float), hipMemcpyDeviceToHost);

    stop_GPU = cpuSecond();

    diferencia_GPU = stop_GPU - start_GPU;

    //Stop GPU part//

    //Start Checking //

    for(int i = 0; i < row3; i++){
        for(int j = 0; j < col3; j++){
            if(fabs(res_F[i * col3 + j] - res[i*col3 + j]) >= error ){
                bien = false;
                printf("Error en: %f %f\n", res_F[i * col3 + j], res[i * col3 + j]);
            }
        }
    }

    if(bien){
        printf("Comparing the output for each implementation.. Correct!\n");
    }else {
        printf("Comparing the output for each implementation.. Incorrect!\n");
    }


    printf("Duration of the CPU: %f\n", diferencia_CPU);
    printf("Duration of the GPU: %f\n", diferencia_GPU);

    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(res);
    hipHostFree(res_F);
    hipFree(A_GPU);
    hipFree(B_GPU);
    hipFree(res_GPU);

}


void matrix_preparation(){

    int m, n, k;

    printf("\n");
    printf("\n");
    printf("Introduce las filas de A:\n");
    fflush(stdout);
    scanf("%d", &m);
    printf("Introduce las columnas de A:\n");
    fflush(stdout);
    scanf("%d", &n);
    printf("Introduce las columnas de B:\n");
    fflush(stdout);
    scanf("%d", &k);

    bool bien = true;


    double start_GPU, stop_GPU;
    double start_CPU, stop_CPU;
    double diferencia_CPU, diferencia_GPU;

    float *A = (float *)malloc(m * n * sizeof(float));
    float *B = (float *)malloc(n * k * sizeof(float));
    float *res = (float *)malloc(m * k * sizeof(float));
    float *res_F = (float *)malloc(m * k * sizeof(float));

    float *A_GPU;
    float *B_GPU;
    float *res_GPU;

    hipMalloc(&A_GPU, m * n * sizeof(float));
    hipMalloc(&B_GPU, n * k * sizeof(float));
    hipMalloc(&res_GPU, m  * k * sizeof(float));
    init_Array(A, m, n);
    init_Array(B, n, k);
    hipMemcpy(A_GPU, A, m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_GPU, B, n * k * sizeof(float), hipMemcpyHostToDevice);

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_colm = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_colm, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    //init CPU part//

    start_CPU = cpuSecond();

    for(int i = 0; i < m; i++){
        for(int j = 0; j < k; j++){
            float cont = 0.0;
            for(int z = 0; z < n; z++){
                cont += A[i * n + z] * B[z * k + j];
            }
            res[i * k + j] = cont;
        }
    }

    stop_CPU = cpuSecond();
    diferencia_CPU = stop_CPU - start_CPU;

    //init GPU Part//

    start_GPU = cpuSecond();

    mmatrix<<<dimGrid, dimBlock>>>(A_GPU, B_GPU, res_GPU, m, n, k);

    hipDeviceSynchronize();
    hipMemcpy(res_F, res_GPU, m * k * sizeof(float), hipMemcpyDeviceToHost);
    
    stop_GPU = cpuSecond();
    diferencia_GPU = stop_GPU - start_GPU;


    //check if it is correct//
    for(int i = 0; i < m; i++){
        for(int j = 0; j < k; j++){
            if(fabs(res_F[i * k + j] - res[i*k + j]) >= error ){
                bien = false;
                break;
            }
        }
        if(!bien){break;}
    }

    if(bien){
        printf("Comparing the output for each implementation.. Correct!\n");
    }else {
        printf("Comparing the output for each implementation.. Incorrect!\n");
    }


    printf("Duration of the CPU: %f\n", diferencia_CPU);
    printf("Duration of the GPU: %f\n", diferencia_GPU);

    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(res);
    hipHostFree(res_F);
    hipFree(A_GPU);
    hipFree(B_GPU);
    hipFree(res_GPU);

}


int main( int argc, char *argv[]){
    
    int op;
    bool salir = false;

 
    START: printf("\n");
    printf("\n");
    printf("******************************************\n");
    printf("* Select the operation you want to do:   *\n");
    printf("*                                        *\n");
    printf("* 1. Matrix Multiplication               *\n");
    printf("* 2. Matrix Convection                   *\n");
    printf("* 3. Matrix Transpose                    *\n");
    printf("* 4. Matrix-Vector Sparse                *\n");
    printf("* 5. Exit                                *\n");
    printf("*                                        *\n");
    printf("******************************************\n");
    fflush(stdout);
    printf(" Introduce the number of the operation:\n");
    fflush(stdout);
    scanf("%d", &op);

    if(op == 0) {

        printf("You dont intriduce a valid options, please do it again.\n");
        goto START;
    }

    switch(op){
        case 1:

            matrix_preparation();
            fflush(stdout);
            break;

        case 2:
        
            conv_preparation();
            fflush(stdout);
            break;

        case 3:

            tranpose_preparation();
            fflush(stdout);
            break;
        
        case 4:

            sparse_preparation();
            fflush(stdout);
            break;

        case 5:

            salir = true;
            printf("Successful Exit\n");
            fflush(stdout);
            break;

        default:
            printf("You dont select any option, please do it again\n");
            fflush(stdout);
            break;
    }
    if(!salir){goto START;}
    return 0;
}
