
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#define error 1e-6

#define NUM_ITERATIONS 1000
#define NUM_PARTICLES 10000
#define BLOCK_SIZE 256
#define NSTREAMS 2

struct particle {
    float position[3];
    float velocity[3];
};

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

__host__ __device__ void uptdateParticle(particle *particula, int iter, int id, int num_p){
    //update the velocity:
    particula[id].velocity[0] = (3*id + iter) % num_p;
    particula[id].velocity[1] = (4*id + iter) % num_p;
    particula[id].velocity[2] = (5*id + iter) % num_p;

    //update the position:
    particula[id].position[0] = particula[id].position[0] + particula[id].velocity[0]; 
    particula[id].position[1] = particula[id].position[1] + particula[id].velocity[1]; 
    particula[id].position[2] = particula[id].position[2] + particula[id].velocity[2]; 
}

__global__ void timeStep(particle *particles, int iter, int num_p, int offset){

    const int id = offset + threadIdx.x + blockIdx.x*blockDim.x;
    if(id < num_p){
        uptdateParticle(particles, iter, id, num_p);
    }
}



int main( int argc, char *argv[]){

    
    bool bien = true;
    


    double start_GPU, stop_GPU;
    double start_CPU, stop_CPU;
    double diferencia_CPU, diferencia_GPU;


    particle *particlesCPU = (particle*)malloc(NUM_PARTICLES * sizeof(particle));
    particle *particlesGPU;
    particle *resCPU;

    hipHostMalloc((void**)&resCPU, NUM_PARTICLES * sizeof(particle), hipHostMallocDefault);
    memset(resCPU,0,NUM_PARTICLES * sizeof(particle));

    const int streamSize = NUM_PARTICLES / NSTREAMS;
    const int StreamBytes = streamSize * sizeof(particle);
    hipStream_t stream[NSTREAMS];
    for(int i = 0; i < NSTREAMS; i++){
        hipStreamCreate(&stream[i]);
    }
    int GRID = (streamSize + BLOCK_SIZE - 1)/BLOCK_SIZE;

    // CPU part//

    start_CPU = cpuSecond();

    for(int i = 0; i < NUM_ITERATIONS; i++){
        for(int j = 0; j < NUM_PARTICLES; j++){
            uptdateParticle(particlesCPU, i, j, NUM_PARTICLES);
        }
    };


    stop_CPU = cpuSecond();
    diferencia_CPU = stop_CPU - start_CPU;

    // Finish CPU part

    //Start GPU part

    start_GPU = cpuSecond();
    hipMalloc((void**)&particlesGPU, NUM_PARTICLES * sizeof(particle));

    for(int s = 0; s < NSTREAMS; s++){
        
        int offset = s * streamSize;
        hipMemcpyAsync(&particlesGPU[offset], &resCPU[offset], StreamBytes, hipMemcpyHostToDevice, stream[s]);
    }

    for(int s = 0; s < NSTREAMS; s++){
        
        int offset = s * streamSize;
        for(int i = 0; i < NUM_ITERATIONS; i++){
            
            timeStep<<<GRID, BLOCK_SIZE, 0, stream[s]>>>(particlesGPU, i, NUM_PARTICLES, offset);
        
        }
        
    }

    for(int s = 0; s < NSTREAMS; s++){
        
        int offset = s * streamSize;
        hipMemcpyAsync(&resCPU[offset], &particlesGPU[offset], StreamBytes, hipMemcpyDeviceToHost, stream[s]);
    }


    
    hipDeviceSynchronize();

    stop_GPU = cpuSecond();

    diferencia_GPU = stop_GPU - start_GPU;

    for(int i = 0; i < NUM_PARTICLES && bien; i++){
        for(int dim = 0; dim < 3; dim++){
            if(abs(particlesCPU[i].position[dim] - resCPU[i].position[dim]) > error ){
                printf("error: %d %d\n", i, dim);
                bien = false;
            }
        }
    }

    printf("NUM_ITERATIONS: %d\n", NUM_ITERATIONS);
    printf("NUM_PARTICLES: %d\n", NUM_PARTICLES);
    printf("BLOCK_SIZE: %d\n", BLOCK_SIZE);
    if(bien){
        printf("datos correctos\n");
    }else{
        printf("datos incorrectos\n");
    }
        
    hipFree(particlesGPU);
    hipHostFree(resCPU);
    delete[] particlesCPU;
    
    for(int i = 0; i < NSTREAMS; i++){
        hipStreamDestroy(stream[i]);
    }
    

    printf("Duration of the CPU: %f\n", diferencia_CPU);
    printf("Duration of the GPU: %f\n", diferencia_GPU);
    printf("--------------------------------------------\n");
    
    return 0;
}