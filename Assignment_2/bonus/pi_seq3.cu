#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <sys/time.h>

#define SEED     921


#define TPB 256
#define NUM_ITER 100000000
#define NUM_THREADS  10000
#define NUM_ITER_THREADS (NUM_ITER/NUM_THREADS)


double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}


__global__ void count_nom(int *d_res, hiprandState *states){
    const int idx = threadIdx.x + blockIdx.x*blockDim.x;
    double x,y,z;
    const int a = 1;

    if (idx >= NUM_THREADS) return;

    const int s_idx = threadIdx.x;
    __shared__ int s_prod[TPB];

    int seed = idx; // different seed per thread
    hiprand_init(seed, idx, 0, &states[idx]);


    for (int iter = 0; iter < NUM_ITER_THREADS; iter++) {
        x = hiprand_uniform (&states[idx]);
        y = hiprand_uniform (&states[idx]);
        z = sqrt((x*x) + (y*y));

        if (z <= 1.0)
        {
            s_prod[s_idx]++;
        }
    }

    if (s_idx == 0) {
    int blockSum = 0;
    for (int j = 0; j < blockDim.x; ++j) {
      blockSum += s_prod[j];
    }
    printf("Block_%d, blockSum = %d\n", blockIdx.x, blockSum);
    // Try each of two versions of adding to the accumulator
    if (ATOMIC) {
      atomicAdd(d_res, blockSum);
    }
  }

}



int main(int argc, char* argv[])
{
    double pi;
    double start_time, stop_time, diference;
    int grid = (NUM_THREADS + TPB - 1)/ TPB;
    
    int *d_res;
    int *count = (int*)malloc(sizeof(int));
    hipMalloc(&d_res, sizeof(int));

    srand(SEED); // Important: Multiply SEED by "rank" when you introduce MPI!
    
    hiprandState *dev_random;
    hipMalloc((void**)&dev_random, grid*TPB*sizeof(hiprandState));

    
    // Calculate PI following a Monte Carlo method
    start_time = cpuSecond();

    count_nom<<<grid, TPB>>>(d_res, dev_random);
    
    hipDeviceSynchronize();

    hipMemcpy(count, d_res,sizeof(int), hipMemcpyDeviceToHost);

    stop_time = cpuSecond();

    diference = stop_time - start_time;

    
    // Estimate Pi and display the result
    pi = ((double)count[0] / (double)(NUM_ITER_THREADS * NUM_THREADS)) * 4.0;
    
    printf("The result is %f\n", pi);
    printf("The execution time is %f\n", diference);
    
    return 0;
}
