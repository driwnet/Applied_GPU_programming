#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define TPB 256
#define ARRAY_SIZE 10000
#define GRID (N + TPB - 1)/TPB
#define error 0.05


__global__ void saxpy(float *x, float *y, const float a){

    const int id = threadIdx.x + blockIdx.x*blockDim.d_x;
    if (id < ARRAY_SIZE){
        y[id] = a*x[id] + y[id];
    }
}

int main(){
    float *x = (float*)malloc(ARRAY_SIZE*sizeof(float));
    float *y = (float*)malloc(ARRAY_SIZE*sizeof(float));
    float res = (float*)malloc(ARRAY_SIZE*sizeof(float));
    const int a = 2;
    bool comp = true;

    float *d_x;
    float *d_y;
    hipMalloc(&d_x, ARRAY_SIZE*sideof(float));
    hipMalloc(&d_y, ARRAY_SIZE*sideof(float));

    for(int i = 0; i < ARRAY_SIZE; i++){
        x[i] = rand() % 1000;
        y[i] = rand() % 1000;
    }

    hipMemcpy(d_x, x, ARRAY_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, ARRAY_SIZE * sizeof(float), hipMemcpyHostToDevice);

    for(int i = 0;i<ARRAY_SIZE,i++){
        res[i] = a*x[i] + y[i];
    }
    printf("Computing SAXPY on the CPU.. Done!\n");
    saxpy<<<GRID, TPB>>>(d_x, d_y, a);
    cudaDeviceSynchtonize();
    
    hipMemcpy(y, d_y, ARRAY_SIZE * sizeof(float), hipMemcpyDeviceToHost);
    printf("Computing SAXPY on the GPU.. Done!\n");

    for( int i = 0; i < N && comp, i++){
        if (abs(res[i] - y[i]) > error){
            comp = false;
        }
    }
    if(comp){
        printf("Comparing the putput for each implementation.. Correct!");
    }else {
        printf("Comparing the putput for each implementation.. Incorrect!");
    }
    free(x);
    free(y);
    hipFree(d_x);
    hipFree(d_y);

    return 0;
}
